#include "hip/hip_runtime.h"
#include "util.h"
#include <math.h>
#include "image.h"
#include "ghost.h"

#define NUM_SHIFTS 30
#define DEFAULT_THRESHOLD 0.15
#define DEFAULT_SQUARE_WIDTH 5
#define DEFAULT_TIMES 32
#define DEFAULT_LINES 10

#define DECLARE_BLOCKS(w, h) \
    const int BLOCKS_WIDTH = ceil_div(width, BLOCK_DIM_SIDE);   \
    const int BLOCKS_HEIGHT = ceil_div(height, BLOCK_DIM_SIDE); \
    const dim3 NUM_BLOCKS = dim3(BLOCKS_WIDTH, BLOCKS_HEIGHT);

#define DECLARE_INDEXES(w, h) \
    int x = threadIdx.x + blockIdx.x * blockDim.x;  \
    int y = threadIdx.y + blockIdx.y * blockDim.y;  \
    if (x < (w) || y < (h))                         \
        return;



// step 1

__device__ int find_edges_left_right(double *brightness,
        int width, int height, int x, int y, double threshold)
{
    double avg_left  = (brightness[idx(x-1, y-1, width, height)]
                     +  brightness[idx(x-1, y  , width, height)]
                     +  brightness[idx(x-1, y+1, width, height)]) / 3.0;
    double avg_right = (brightness[idx(x+1, y-1, width, height)]
                     +  brightness[idx(x+1, y  , width, height)]
                     +  brightness[idx(x+1, y+1, width, height)]) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall,
                                                0.0, 1.0);
}

__device__ int find_edges_top_bottom(double *brightness,
        int width, int height, int x, int y, double threshold)
{
    double avg_left  = (brightness[idx(x-1, y-1, width, height)]
                     +  brightness[idx(x  , y-1, width, height)]
                     +  brightness[idx(x+1, y-1, width, height)]) / 3.0;
    double avg_right = (brightness[idx(x-1, y+1, width, height)]
                     +  brightness[idx(x  , y+1, width, height)]
                     +  brightness[idx(x+1, y+1, width, height)]) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall,
                                                0.0, 1.0);
}

__device__ int find_edges_upleft_downright(double *brightness,
        int height, int width, int x, int y, double threshold)
{
    double avg_left  = (brightness[idx(x-1, y-1, width, height)]
                     +  brightness[idx(x  , y-1, width, height)]
                     +  brightness[idx(x-1, y  , width, height)]) / 3.0;
    double avg_right = (brightness[idx(x+1, y  , width, height)]
                     +  brightness[idx(x  , y+1, width, height)]
                     +  brightness[idx(x+1, y+1, width, height)]) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall,
                                                0.0, 1.0);
}

__device__ int find_edges_downleft_upright(double *brightness,
        int height, int width, int x, int y, double threshold)
{
    double avg_left  = (brightness[idx(x-1, y+1, width, height)]
                     +  brightness[idx(x  , y+1, width, height)]
                     +  brightness[idx(x-1, y  , width, height)]) / 3.0;
    double avg_right = (brightness[idx(x  , y-1, width, height)]
                     +  brightness[idx(x+1, y-1, width, height)]
                     +  brightness[idx(x+1, y  , width, height)]) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall,
                                                0.0, 1.0);
}

__global__ void find_all_edges(double *brightness, int w, int h,
        double threshold, u8 *edges)
{
    DECLARE_INDEXES(w, h)
    edges[IDX(x, y, w)] =
              find_edges_left_right(brightness, w, h, x, y, threshold)
     ||       find_edges_top_bottom(brightness, w, h, x, y, threshold)
     || find_edges_upleft_downright(brightness, w, h, x, y, threshold)
     || find_edges_downleft_upright(brightness, w, h, x, y, threshold);
}



// step 2

__device__ u8 *matches[NUM_SHIFTS];

void allocate_matches(int width, int height)
{
    u8 *tmp[NUM_SHIFTS];
    for (int i = 0; i < NUM_SHIFTS; i++)
        tmp[i] = ALLOCATE_GPU(u8, width * height);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(matches), tmp, sizeof(tmp)));
}

void write_matches(int width, int height)
{
#ifndef NO_WRITES
    u8 *tmp[NUM_SHIFTS];
    checkCudaErrors(hipMemcpyFromSymbol(tmp, HIP_SYMBOL(matches), sizeof(tmp)));
    for (int i = 0; i < NUM_SHIFTS; i++)
        write_gpu_image(tmp[i], width, height, 0, IMTYPE_BINARY, make_filename("matches", PAR, i));
#endif
}

void free_matches()
{
    u8 *tmp[NUM_SHIFTS];
    checkCudaErrors(hipMemcpyFromSymbol(tmp, HIP_SYMBOL(matches), sizeof(tmp)));
    for (int i = 0; i < NUM_SHIFTS; i++)
        checkCudaErrors(hipFree(tmp[i]));
}

// this function records the edge-pixel match-ups at every shift
__global__ void fillup_matches(u8 *left_edges, u8 *right_edges,
        int width, int height)
{
    DECLARE_INDEXES(width, height)
    int index = IDX(x, y, width);
    for (int i = 0; i < NUM_SHIFTS; i++) {
        int shift = idx(x+i, y, width, height);
        // ^ the +i accomplishes the sliding process
        matches[i][index] = left_edges[index] == right_edges[shift];
    }
}



// step 3

// the square for each pixel is to be centered on that pixel.
// the double for loop is slightly different than the original,
// going from -half to +half.
__global__ void addup_pixels_in_square(int i, int width, int height,
        int square_width, i32 *total)
{
    DECLARE_INDEXES(width, height)
    u8 *pixels = matches[i];
    int cur = IDX(x, y, width);
    int half = square_width / 2;
    for (int sy = -half; sy <= half; sy++) {
        for (int sx = -half; sx <= half; sx++) {
            int rel = idx(x + sx, y + sy, width, height);
            total[cur] += (i32) pixels[rel];
        }
    }
}

__device__ i32 *scores[NUM_SHIFTS];

void allocate_scores(int width, int height)
{
    i32 *tmp[NUM_SHIFTS];
    for (int i = 0; i < NUM_SHIFTS; i++)
        tmp[i] = ALLOCATE_GPU(i32, width * height);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(scores), tmp, sizeof(tmp)));
}

void write_scores(int width, int height)
{
#ifndef NO_WRITES
    i32 *tmp[NUM_SHIFTS];
    checkCudaErrors(hipMemcpyFromSymbol(tmp, HIP_SYMBOL(scores), sizeof(tmp)));
    for (int i = 0; i < NUM_SHIFTS; i++)
        write_gpu_image(tmp[i], width, height, 0, IMTYPE_GRAY_INT, make_filename("scores", PAR, i));
#endif
}

void free_scores()
{
    i32 *tmp[NUM_SHIFTS];
    checkCudaErrors(hipMemcpyFromSymbol(tmp, HIP_SYMBOL(scores), sizeof(tmp)));
    for (int i = 0; i < NUM_SHIFTS; i++)
        checkCudaErrors(hipFree(tmp[i]));
}

__global__ void record_score(int i, i32 *sum, int width, int height)
{
    DECLARE_INDEXES(width, height)
    int index = IDX(x, y, width);
    // record a score whenever there was a match-up
    if (matches[i][index] == 1)
        scores[i][index] = sum[index];
}

void fillup_scores(int width, int height, int square_width, i32 *sum)
{
    DECLARE_BLOCKS(width, height)
    for (int i = 0; i < NUM_SHIFTS; i++) {
        hipMemset(sum, 0, sizeof(sum[0]) * width * height);
        addup_pixels_in_square<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(
            i, width, height, square_width, sum
        );
        write_gpu_image(sum, width, height, 0, IMTYPE_GRAY_INT, make_filename("score_all", PAR, i));
        record_score<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(
            i, sum, width, height
        );
    }
}

// this function computes the web of known shifts. recall that
// the shift at each pixel corresponds directly to the elevation.
__global__ void find_highest_scoring_shifts(i32 *best_scores,
        i32 *winning_shifts, int width, int height)
{
    DECLARE_INDEXES(width, height)
    int index = IDX(x, y, width);
    // the following for loop makes sure that each pixel in the
    // 'best_scores' variable contains the maximum score found at any shift.
    for (int i = 0; i < NUM_SHIFTS; i++)
        best_scores[index] = MAX(scores[i][index], best_scores[index]
    // the following for loop records a "winning"
    // shift at every pixel whose score is the best.
    for (int i = 0; i < NUM_SHIFTS; i++)
        if (scores[i][index] == best_scores[index])
            winning_shifts[index] = i+1;
}



// step 4

__global__ void fill_web_holes_step(i32 *web, i32 *tmp, int width)
{
    DECLARE_INDEXES(width, height)
    if (tmp[IDX(x, y, width)] == 0)
        web[IDX(x, y, width)] =
            (tmp[IDX(x+1, y,   width)]  // neighbor to the right
           + tmp[IDX(x,   y+1, width)]  // neighbor above
           + tmp[IDX(x-1, y,   width)]  // neighbor to the left
           + tmp[IDX(x,   y-1, width)]) // neighbor below
           / 4;
}

// each time though the loop, every pixel not on the web (i.e., every pixel that is not
// zero to begin with) takes on the average elevation of its four neighbors. therefore,
// the web pixels gradually "spread" their elevations across the holes, while they
// themselves remain unchanged.
i32 *fill_web_holes(i32 *web, i32 *tmp, int width, int height, int times)
{
    DECLARE_BLOCKS(width, height)
    for (int i = 0; i < times; i++) {
        fill_web_holes_step<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(web, tmp, width);
        SWAP(i32 *, web, tmp);
    }
    // this is now a more or less smooth surface.
    return web;
}

i32 image_max(i32 *im, int width, int height) { return array_max_gpu(im, width*height); }
i32 image_min(i32 *im, int width, int height) { return array_min_gpu(im, width*height); }

__global__ void draw_contour_map_kernel(i32 *web, int width,
        int num_lines, i32 max_elevation, i32 min_elevation, u8 *out)
{
    // the idea is to divide the whole range of elevations into a number
    // of intervals, then to draw a contour line at every interval.
    DECLARE_INDEXES(width, height)
    int i = IDX(x, y, width);
    // the variable 'interval' tells us how many
    // elevations, or shifts, to skip between contour lines.
    i32 range    = max_elevation - min_elevation,
        interval = range / num_lines;
    // this line draws all the elevation contours at once.
    out[i] = ((web[i] - min_elevation) % interval) == 0;
}

void draw_contour_map(i32 *web, int width, int height,
        int num_lines, u8 *out)
{
    DECLARE_BLOCKS(width, height)
    i32 immax = image_max(web, width, height),
        immin = image_min(web, width, height);
    draw_contour_map_kernel<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(
        web, width, num_lines, immax, immin, out
    );
}



typedef struct AlgorithmParams {
    double threshold;
    int square_width;
    int times;
    int lines_to_draw;
} AlgorithmParams;

void algorithm(double *first, double *second, int width, int height, AlgorithmParams params)
{
    DECLARE_BLOCKS(width, height)
    u8 *first_edges  = ALLOCATE_GPU(u8, width * height),
       *second_edges = ALLOCATE_GPU(u8, width * height);
    i32 *buf         = ALLOCATE_GPU(i32, width * height);
    i32 *web         = ALLOCATE_GPU(i32, width * height);
    i32 *tmp         = ALLOCATE_GPU(i32, width * height);
    u8 *out          = ALLOCATE_GPU(u8, width * height);
    allocate_matches(width, height);
    allocate_scores(width, height);

    double t1 = get_time();

    // first step: find edges in both images
    find_all_edges<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(first,  width, height, params.threshold, first_edges);
    find_all_edges<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(second, width, height, params.threshold, second_edges);
    write_gpu_image(first_edges,  width, height, 0, IMTYPE_BINARY, make_filename("edges", PAR, 1));
    write_gpu_image(second_edges, width, height, 0, IMTYPE_BINARY, make_filename("edges", PAR, 2));

    // second step: match edges between images
    fillup_matches<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(first_edges, second_edges, width, height);
    write_matches(width, height);

    fillup_scores(width, height, params.square_width, buf);
    write_scores(width, height);
    hipMemset(buf, 0, sizeof(buf[0]) * width * height);
    find_highest_scoring_shifts<<<NUM_BLOCKS, BLOCK_DIM_2D>>>(buf, web, width, height);
    write_gpu_image(buf, width, height, 0, IMTYPE_GRAY_INT, make_filename("score_best", PAR, 0));
    write_gpu_image(web, width, height, 0, IMTYPE_GRAY_INT, make_filename("web", PAR, 1));

    // third step: draw contour lines
    web = fill_web_holes(web, tmp, width, height, params.times);
    checkCudaErrors(hipMemcpy(tmp, web, sizeof(web[0]) * width * height, hipMemcpyDeviceToDevice));
    write_gpu_image(web, width, height, 0, IMTYPE_GRAY_INT, make_filename("web", PAR, 2));
    draw_contour_map(web, width, height, params.lines_to_draw, out);
    write_gpu_image(out, width, height, 0, IMTYPE_BINARY, make_filename("output", PAR, 0));

    hipDeviceSynchronize();
    double t2 = get_time();
    double elapsed = t2 - t1;
    printf("width = %d, height = %d, t1 = %f, t2 = %f, elapsed = %f\n", width, height, t1, t2, elapsed);

    checkCudaErrors(hipFree(first_edges));
    checkCudaErrors(hipFree(second_edges));
    checkCudaErrors(hipFree(web));
    checkCudaErrors(hipFree(out));
    checkCudaErrors(hipFree(buf));
    checkCudaErrors(hipFree(tmp));
    free_matches();
    free_scores();
}


int main(int argc, char *argv[])
{
   if (argc < 3) {
        fprintf(stderr, "usage: stereomatch [image 1] [image 2] [threshold = %g] "
                        "[square_width = %d] [times = %d] [lines = %d]\n",
                        DEFAULT_THRESHOLD, DEFAULT_SQUARE_WIDTH, DEFAULT_TIMES, DEFAULT_LINES);
        return 1;
    }

    Image first, second;
    if (read_image(argv[1], &first))
        return 1;
    if (read_image(argv[2], &second))
        return 1;
    if (first.width != second.width || first.height != second.height) {
        fprintf(stderr, "error: the two images must have equal width and height\n");
        return 1;
    }

    AlgorithmParams params = {
        .threshold     = DEFAULT_THRESHOLD,
        .square_width  = DEFAULT_SQUARE_WIDTH,
        .times         = DEFAULT_TIMES,
        .lines_to_draw = DEFAULT_LINES
    };

    if (argc >= 4 && parse_double(argv[3], &params.threshold)) {
        fprintf(stderr, "error: threshold must be a number\n");
        return 1;
    }
    if (argc >= 5 && parse_int(argv[4], &params.square_width)) {
        fprintf(stderr, "error: square_width must be a number\n");
        return 1;
    }
    if (argc >= 6 && parse_int(argv[5], &params.times)) {
        fprintf(stderr, "error: times must be a number\n");
        return 1;
    }
    if (argc >= 7 && parse_int(argv[6], &params.lines_to_draw)) {
        fprintf(stderr, "error: lines must be a number\n");
        return 1;
    }

    if (params.threshold < 0.0 || params.threshold > 1.0) {
        fprintf(stderr, "error: threshold must be between 0 and 1\n");
        return 1;
    }
    if (params.square_width > first.width || params.square_width > first.height) {
        fprintf(stderr, "error: square width must not be higher than image width/height\n");
        return 1;
    }

    double *first_gpu  = MAKE_GPU_COPY(double, first.data,  first.width * first.height),
           *second_gpu = MAKE_GPU_COPY(double, second.data, first.width * first.height);
    algorithm(first_gpu, second_gpu, first.width, first.height, params);

    free(first.data);
    free(second.data);
    return 0;
}

