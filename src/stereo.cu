#include "hip/hip_runtime.h"
#include "util.h"
#include <math.h>
#include "image.h"

#define NUM_SHIFTS 30
#define DEFAULT_THRESHOLD 0.15
#define DEFAULT_SQUARE_WIDTH 5
#define DEFAULT_TIMES 32
#define DEFAULT_LINES 10

// assume maximum thread no is 1024 (32*32)
#define BLOCK_DIM 32



// step 1

int __device__ find_edges_left_right(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y-1, width)];
    double v2 = brightness[idx(x-1, y  , width)];
    double v3 = brightness[idx(x-1, y+1, width)];
    double v4 = brightness[idx(x+1, y-1, width)];
    double v5 = brightness[idx(x+1, y  , width)];
    double v6 = brightness[idx(x+1, y+1, width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

int __device__ find_edges_top_bottom(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y-1, width)];
    double v2 = brightness[idx(x  , y-1, width)];
    double v3 = brightness[idx(x+1, y-1, width)];
    double v4 = brightness[idx(x-1, y+1, width)];
    double v5 = brightness[idx(x  , y+1, width)];
    double v6 = brightness[idx(x+1, y+1, width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

int __device__ find_edges_upleft_downright(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y-1, width)];
    double v2 = brightness[idx(x  , y-1, width)];
    double v3 = brightness[idx(x-1, y  , width)];
    double v4 = brightness[idx(x+1, y  , width)];
    double v5 = brightness[idx(x  , y+1, width)];
    double v6 = brightness[idx(x+1, y+1, width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

int __device__ find_edges_downleft_upright(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y+1, width)];
    double v2 = brightness[idx(x  , y+1, width)];
    double v3 = brightness[idx(x-1, y  , width)];
    double v4 = brightness[idx(x  , y-1, width)];
    double v5 = brightness[idx(x+1, y-1, width)];
    double v6 = brightness[idx(x+1, y  , width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

void __global__ find_all_edges(double *brightness, u8 *edges, int width, int height, double threshold)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    edges[IDX(x, y, width)] =
        find_edges_left_right(brightness, width, x, y, threshold)
     || find_edges_top_bottom(brightness, width, x, y, threshold)
     || find_edges_upleft_downright(brightness, width, x, y, threshold)
     || find_edges_downleft_upright(brightness, width, x, y, threshold);
}



// step 2

// a WxH size array used to keep matches
u8 __device__ *matches[NUM_SHIFTS];

void allocate_matches(int width, int height)
{
    void *tmp[NUM_SHIFTS];
    for (int i = 0; i < NUM_SHIFTS; i++)
        hipMalloc(&tmp[i], width * height * sizeof(matches[0]));
    hipMemcpyToSymbol(HIP_SYMBOL(matches), tmp, sizeof(tmp));
}

void write_matches(int width, int height)
{
    u8 *tmp[NUM_SHIFTS];
    hipMemcpyFromSymbol(tmp, HIP_SYMBOL(matches), sizeof(tmp));
    for (int i = 0; i < NUM_SHIFTS; i++)
        write_image_from_gpu(tmp[i], width, height, 0, IMTYPE_BINARY, "matches", i);
}

void __global__ fillup_matches(u8 *left_edges, u8 *right_edges, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int i = 0; i < NUM_SHIFTS; i++) {
        int index = idx(x,   y, width),
            shift = idx(x+i, y, width);
        // ^ the +i accomplishes the sliding process
        matches[i][index] = left_edges[index] == right_edges[shift];
    }
}



typedef struct AlgorithmParams {
    double threshold;
    int square_width;
    int times;
    int lines_to_draw;
} AlgorithmParams;

void algorithm(double *first, double *second, int width, int height, AlgorithmParams params)
{
    const int num_blocks_side = ceil_div(width, BLOCK_DIM);
    const dim3 num_blocks = dim3(num_blocks_side, num_blocks_side);
    const dim3 block_dim  = dim3(BLOCK_DIM, BLOCK_DIM);

    // first step: find edges in both images
    u8 *first_edges; hipMalloc(&first_edges, width * height * sizeof(u8));
    find_all_edges<<<num_blocks, block_dim>>>(first, first_edges, width, height, params.threshold);
    write_image_from_gpu(first_edges, width, height, 0, IMTYPE_BINARY, "edges", 1);

    u8 *second_edges; hipMalloc(&second_edges, width * height * sizeof(u8));
    find_all_edges<<<num_blocks, block_dim>>>(second, second_edges, width, height, params.threshold);
    write_image_from_gpu(second_edges, width, height, 0, IMTYPE_BINARY, "edges", 2);

    // second step: match edges between images
    allocate_matches(width, height);
    fillup_matches<<<num_blocks, block_dim>>>(first_edges, second_edges, width, height);
    write_matches(width, height);
}


int main(int argc, char *argv[])
{
   if (argc < 3) {
        fprintf(stderr, "usage: stereomatch [image 1] [image 2] [threshold = %g] "
                        "[square_width = %d] [times = %d] [lines = %d]\n",
                        DEFAULT_THRESHOLD, DEFAULT_SQUARE_WIDTH, DEFAULT_TIMES, DEFAULT_LINES);
        return 1;
    }

    Image first, second;
    if (read_image(argv[1], &first))
        return 1;
    if (read_image(argv[2], &second))
        return 1;
    if (first.width != second.width || first.height != second.height) {
        fprintf(stderr, "error: the two images must have equal width and height\n");
        return 1;
    }

    AlgorithmParams params = {
        .threshold     = DEFAULT_THRESHOLD,
        .square_width  = DEFAULT_SQUARE_WIDTH,
        .times         = DEFAULT_TIMES,
        .lines_to_draw = DEFAULT_LINES
    };

    if (argc >= 4 && parse_double(argv[3], &params.threshold)) {
        fprintf(stderr, "error: threshold must be a number\n");
        return 1;
    }
    if (argc >= 5 && parse_int(argv[4], &params.square_width)) {
        fprintf(stderr, "error: square_width must be a number\n");
        return 1;
    }
    if (argc >= 6 && parse_int(argv[5], &params.times)) {
        fprintf(stderr, "error: times must be a number\n");
        return 1;
    }
    if (argc >= 7 && parse_int(argv[6], &params.lines_to_draw)) {
        fprintf(stderr, "error: lines must be a number\n");
        return 1;
    }

    double *first_img;
    hipMalloc(&first_img,  first.width  * first.height  * sizeof(double));
    hipMemcpy(first_img, first.data, first.width * first.height * sizeof(double), hipMemcpyHostToDevice);

    double *second_img;
    hipMalloc(&second_img, second.width * second.height * sizeof(double));
    hipMemcpy(second_img, second.data, second.width * second.height * sizeof(double), hipMemcpyHostToDevice);

    algorithm(first_img, second_img, first.width, first.height, params);

    hipDeviceSynchronize();

    return 0;
}
