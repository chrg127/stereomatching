#include "hip/hip_runtime.h"
#include "util.h"
#include <math.h>
#include "image.h"

#define NUM_SHIFTS 30
#define DEFAULT_THRESHOLD 0.15
#define DEFAULT_SQUARE_WIDTH 5
#define DEFAULT_TIMES 32
#define DEFAULT_LINES 10

// assume maximum thread no is 1024 (32*32)
#define BLOCK_DIM 32



// step 1

int __device__ find_edges_left_right(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y-1, width)];
    double v2 = brightness[idx(x-1, y  , width)];
    double v3 = brightness[idx(x-1, y+1, width)];
    double v4 = brightness[idx(x+1, y-1, width)];
    double v5 = brightness[idx(x+1, y  , width)];
    double v6 = brightness[idx(x+1, y+1, width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

int __device__ find_edges_top_bottom(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y-1, width)];
    double v2 = brightness[idx(x  , y-1, width)];
    double v3 = brightness[idx(x+1, y-1, width)];
    double v4 = brightness[idx(x-1, y+1, width)];
    double v5 = brightness[idx(x  , y+1, width)];
    double v6 = brightness[idx(x+1, y+1, width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

int __device__ find_edges_upleft_downright(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y-1, width)];
    double v2 = brightness[idx(x  , y-1, width)];
    double v3 = brightness[idx(x-1, y  , width)];
    double v4 = brightness[idx(x+1, y  , width)];
    double v5 = brightness[idx(x  , y+1, width)];
    double v6 = brightness[idx(x+1, y+1, width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

int __device__ find_edges_downleft_upright(double *brightness, int width, int x, int y, double threshold)
{
    double v1 = brightness[idx(x-1, y+1, width)];
    double v2 = brightness[idx(x  , y+1, width)];
    double v3 = brightness[idx(x-1, y  , width)];
    double v4 = brightness[idx(x  , y-1, width)];
    double v5 = brightness[idx(x+1, y-1, width)];
    double v6 = brightness[idx(x+1, y  , width)];
    double avg_left  = (v1 + v2 + v3) / 3.0;
    double avg_right = (v4 + v5 + v6) / 3.0;
    double overall   = (avg_left + avg_right) / 2.0;
    return fabs(avg_left - avg_right) > CLAMP(threshold * overall, 0.0, 1.0);
}

void __global__ find_all_edges(double *brightness, u8 *edges, int width, int height, double threshold)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    edges[IDX(x, y, width)] =
        find_edges_left_right(brightness, width, x, y, threshold)
     || find_edges_top_bottom(brightness, width, x, y, threshold)
     || find_edges_upleft_downright(brightness, width, x, y, threshold)
     || find_edges_downleft_upright(brightness, width, x, y, threshold);
}



// step 2

// a WxH size array used to keep matches
u8 __device__ *matches[NUM_SHIFTS];

void allocate_matches(int width, int height)
{
    void *tmp[NUM_SHIFTS];
    for (int i = 0; i < NUM_SHIFTS; i++)
        hipMalloc(&tmp[i], width * height * sizeof(matches[0]));
    hipMemcpyToSymbol(HIP_SYMBOL(matches), tmp, sizeof(tmp));
}

void write_matches(int width, int height)
{
    u8 *tmp[NUM_SHIFTS];
    hipMemcpyFromSymbol(tmp, HIP_SYMBOL(matches), sizeof(tmp));
    for (int i = 0; i < NUM_SHIFTS; i++)
        write_image_from_gpu(tmp[i], width, height, IMTYPE_BINARY, "matches", i);
}

void __global__ fillup_matches(u8 *left_edges, u8 *right_edges, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int i = 0; i < NUM_SHIFTS; i++) {
        int index = idx(x,   y, width),
            shift = idx(x+i, y, width);
        // ^ the +i accomplishes the sliding process
        matches[i][index] = left_edges[index] == right_edges[shift];
    }
}

/*
// a WxH size array used to keep scores
i32 *scores[NUM_SHIFTS];

void allocate_scores(int width, int height)
{
    for (int i = 0; i < NUM_SHIFTS; i++)
        scores[i] = (i32 *) xmalloc(width * height, sizeof(scores[0]));
}

 * for each pixel in parallel:
 *
 * *.. .*. ..* ... ... ... ... ... ...
 * ... ... ... *.. .*. ..* ... ... ...
 * ... ... ... ... ... ... *.. .*. ..*
 *
 * (where the considered pixel is at the center and square_width = 3)
 * pixels must be a binary image.
void addup_pixels_in_square(u8 *pixels, int width, int height, int square_width, i32 *total)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int half = square_width / 2;
    memset(total, 0, sizeof(total[0]) * width * height);
    for (int sy = 0; sy < square_width; sy++) {
        for (int sx = 0; sx < square_width; sx++) {
            total[idx(x, y, width)] +=
                (i32) pixels[idx(x + sx - half,
                                 y + sy - half, width)];
        }
    }
}

void fillup_scores(int width, int height, int square_width, i32 *sum)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int i = 0; i < NUM_SHIFTS; i++) {
        addup_pixels_in_square(matches[i], width, height, square_width, sum);
        int index = idx(x, y, width);
        // record a score whenever there was a match-up
        if (matches[i][index] == 1)
            scores[i][index] = sum[index];
    }
}

// this function computes the web of known shifts. recall that
// the shift at each pixel corresponds directly to the elevation.
void find_highest_scoring_shifts(i32 *best_scores, i32 *winning_shifts, int width, int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    memset(best_scores,    0, sizeof(best_scores[0])    * width * height);
    memset(winning_shifts, 0, sizeof(winning_shifts[0]) * width * height);
    // the following loop makes sure that each pixel in the best_scores
    // image contains the maximum score found at any shift.
    for (int i = 0; i < NUM_SHIFTS; i++) {
        int index = idx(x, y, width);
        if (scores[i][index] > best_scores[index])
            best_scores[index] = scores[i][index];
    }
    write_image(best_scores, width, height, IMTYPE_GRAY_INT, "score_best", 0);
    // the following loop records a 'winning' shift at every pixel
    // whose score is the best.
    for (int i = 0; i < NUM_SHIFTS; i++) {
        int index = idx(x, y, width);
        if (scores[i][index] == best_scores[index])
            winning_shifts[index] = i+1;
    }
}

void __global__ fill_web_holes(i32 *web, i32 *tmp, int width, int times)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    for (int i = 0; i < times; i++) {
        if (web[idx(x, y, width)] == 0) {
            web[idx(x, y, width)] =
                (tmp[idx(x+1, y,   width)]
               + tmp[idx(x,   y+1, width)]
               + tmp[idx(x-1, y,   width)]
               + tmp[idx(x,   y-1, width)]) / 4;
        }
    }
}

i32 *fill_web_holes(i32 *web, int width, int height, int times)
{
    // each time though the loop, every pixel not on the web (i.e., every pixel that is not
    // zero to begin with) takes on the average elevation of its four neighbors. therefore,
    // the web pixels gradually "spread" their elevations across the holes, while they
    // themselves remain unchanged.
    i32 *tmp; hipMalloc(&tmp, width * height, sizeof(web[0]));
    hipMemcpy(tmp, web, sizeof(web[0]) * width * height, hipMemcpyDeviceToDevice);
    for (int i = 0; i < times; i++) {
        fill_web_holes<<<>>>(web, tmp, width, times);
        SWAP(web, tmp, i32 *);
    }
    hipFree(tmp);
    return web;
}



i32 image_max(i32 *im, int width, int height)
{
    i32 max = 0;
    for (int i = 0; i < width*height; i++)
        max = MAX(im[i], max);
    return max;
}

i32 image_min(i32 *im, int width, int height)
{
    i32 min = 0;
    for (int i = 0; i < width*height; i++)
        min = MIN(im[i], min);
    return min;
}

void draw_contour_map(i32 *web, int width, int height, int num_lines, u8 *image_output)
{
    // the idea is to divide the whole range of elevations into a number of intervals,
    // then to draw a contour line at every interval.
    i32 max_elevation = image_max(web, width, height),
        min_elevation = image_min(web, width, height),
        range         = max_elevation - min_elevation,
        interval      = range / num_lines;
    // now the variable 'interval' tells us how many elevations, or shifts, to skip between
    // contour lines.
    for (int y = 0; y < height; y++) {
        for (int x = 0; x < width; x++) {
            image_output[idx(x, y, width)] =
                ((web[idx(x, y, width)] - min_elevation) % interval) == 0;
        }
    }
}

*/



typedef struct AlgorithmParams {
    double threshold;
    int square_width;
    int times;
    int lines_to_draw;
} AlgorithmParams;

void algorithm(double *first, double *second, int width, int height, AlgorithmParams params)
{
    const int num_blocks_side = ceil_div(width, BLOCK_DIM);
    const dim3 num_blocks = dim3(num_blocks_side, num_blocks_side);
    const dim3 block_dim  = dim3(BLOCK_DIM, BLOCK_DIM);

    // first step: find edges in both images
    u8 *first_edges; hipMalloc(&first_edges, width * height * sizeof(u8));
    find_all_edges<<<num_blocks, block_dim>>>(first, first_edges, width, height, params.threshold);
    write_image_from_gpu(first_edges, width, height, IMTYPE_BINARY, "edges", 1);

    u8 *second_edges; hipMalloc(&second_edges, width * height * sizeof(u8));
    find_all_edges<<<num_blocks, block_dim>>>(second, second_edges, width, height, params.threshold);
    write_image_from_gpu(second_edges, width, height, IMTYPE_BINARY, "edges", 2);

    // second step: match edges between images
    allocate_matches(width, height);
    fillup_matches<<<num_blocks, block_dim>>>(first_edges, second_edges, width, height);
    write_matches(width, height);

    /*
    // third step: compute scores for each pixel
    i32 *buf            = (i32 *) xmalloc(width * height, sizeof(i32)),
        *winning_shifts = (i32 *) xmalloc(width * height, sizeof(i32));
    allocate_scores(width, height);
    fillup_scores(width, height, 5, buf);
    find_highest_scoring_shifts(buf, winning_shifts, width, height);
    write_image_from_gpu(winning_shifts, width, height, IMTYPE_GRAY_INT, "web", 1);

    // fourth step: draw contour lines
    i32 *web = winning_shifts;
    u8 *out = (u8 *) xmalloc(width * height, sizeof(u8));
    web = fill_web_holes(web, width, height, times);
    write_image_from_gpu(web, width, height, IMTYPE_GRAY_INT, "web", 2);
    draw_contour_map(web, width, height, lines_to_draw, out);
    write_image_from_gpu(out, width, height, IMTYPE_BINARY, "output", 0);
    */
}


int main(int argc, char *argv[])
{
   if (argc < 3) {
        fprintf(stderr, "usage: stereomatch [image 1] [image 2] [threshold = %g] "
                        "[square_width = %d] [times = %d] [lines = %d]\n",
                        DEFAULT_THRESHOLD, DEFAULT_SQUARE_WIDTH, DEFAULT_TIMES, DEFAULT_LINES);
        return 1;
    }

    Image first, second;
    if (read_image(argv[1], &first))
        return 1;
    if (read_image(argv[2], &second))
        return 1;
    if (first.width != second.width || first.height != second.height) {
        fprintf(stderr, "error: the two images must have equal width and height\n");
        return 1;
    }

    AlgorithmParams params = {
        .threshold     = DEFAULT_THRESHOLD,
        .square_width  = DEFAULT_SQUARE_WIDTH,
        .times         = DEFAULT_TIMES,
        .lines_to_draw = DEFAULT_LINES
    };

    if (argc >= 4 && parse_double(argv[3], &params.threshold)) {
        fprintf(stderr, "error: threshold must be a number\n");
        return 1;
    }
    if (argc >= 5 && parse_int(argv[4], &params.square_width)) {
        fprintf(stderr, "error: square_width must be a number\n");
        return 1;
    }
    if (argc >= 6 && parse_int(argv[5], &params.times)) {
        fprintf(stderr, "error: times must be a number\n");
        return 1;
    }
    if (argc >= 7 && parse_int(argv[6], &params.lines_to_draw)) {
        fprintf(stderr, "error: lines must be a number\n");
        return 1;
    }

    double *first_img;
    hipMalloc(&first_img,  first.width  * first.height  * sizeof(double));
    hipMemcpy(first_img, first.data, first.width * first.height * sizeof(double), hipMemcpyHostToDevice);

    double *second_img;
    hipMalloc(&second_img, second.width * second.height * sizeof(double));
    hipMemcpy(second_img, second.data, second.width * second.height * sizeof(double), hipMemcpyHostToDevice);

    algorithm(first_img, second_img, first.width, first.height, params);

    hipDeviceSynchronize();

    return 0;
}
